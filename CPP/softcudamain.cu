#include "hip/hip_runtime.h"
#include "common.h"
#include "kernels.cu"
#include <eigen3/Eigen/Sparse>


using namespace cv;
using namespace Eigen;

//#define WINSIZE 1
//const int nebsize=(WINSIZE*2+1)*(WINSIZE*2+1);
//tudo isso ja foi definido em kernels.cpp

//se aumentar o tamanho de WINSIZE em faltar memoria pra iniciar um dos kernels, tem que diminuir esse valor
#define MAX_WARPS 16

double *dev_img, *dev_vals, *h_alpha_mtrx, *img, *img_scr, *h_vals;
int *dev_row_inds, *dev_col_inds, *n_mp_dev, *h_row_inds, *h_col_inds;
double lambda;
int tlen;
int n_vals;



void init_vars(int img_w, int img_h){
    tlen=(img_w-2*WINSIZE)*(img_h-2*WINSIZE)*nebsize*nebsize;
        n_vals=0;
    int size_vec_img=img_w*img_h*sizeof(double);
    img=(double*)malloc(3*size_vec_img);
    img_scr=(double*)malloc(size_vec_img);
    hipMalloc((void **)&dev_img, 3*size_vec_img);
    hipMalloc((void **)&dev_row_inds, tlen*sizeof(int));
    hipMalloc((void **)&dev_col_inds, tlen*sizeof(int));
    hipMalloc((void **)&dev_vals, tlen*sizeof(double));
    int dist_h, dist_w;
    for(int i=0; i<img_w*img_h; i++){
        dist_h=min(2*WINSIZE,min(i%img_h,img_h-1-i%img_h));
        dist_w=min(2*WINSIZE,min(i/img_h,img_w-1-i/img_h));
        n_vals+=(1+2*WINSIZE+dist_h)*(1+2*WINSIZE+dist_w);
    }
    h_row_inds=(int*)malloc(tlen*sizeof(int));
    h_col_inds=(int*)malloc(tlen*sizeof(int));
    h_vals=(double*)malloc(tlen*sizeof(double));

    int n_dev;
    hipGetDeviceCount(&n_dev);
    n_mp_dev=(int*)malloc(n_dev*sizeof(int));
    hipDeviceProp_t dev_prop[n_dev];
    hipSetDevice(0);
    for(int i=0; i<n_dev; i++){
            hipGetDeviceProperties(&dev_prop[i], i);
        n_mp_dev[i]=dev_prop[i].multiProcessorCount;
    }
}

void free_vars(){
    free(img);
    free(img_scr);
    hipFree(dev_col_inds);
    hipFree(dev_vals);
    hipFree(dev_row_inds);
    hipFree(dev_img);
}

Mat solve_alpha(const Mat *img_cv, const Mat *img_scr_cv){
    int c0, c1, c2;
    int img_w=img_cv->cols;
    int img_h=img_cv->rows;

    if(img_cv->type()==CV_8UC3){
        //se a imagem e colorida inverte os canais para converter de BGR para RGB
        c0=2;
        c1=1;
        c2=0;
    } else if(img_cv->type()==CV_8UC1){
        //se e monocromatica ja converte pra colorida
        c0=0;
        c1=0;
        c2=0;
    }
#pragma omp parallel for
    for(int i=0; i<img_w; i++){
        for(int j=0;j<img_h; j++){
            img[3*(i*img_h+j)]=img_cv->at<Vec3b>(j,i)[c0]/255.0l;
            img[3*(i*img_h+j)+1]=img_cv->at<Vec3b>(j,i)[c1]/255.0l;
            img[3*(i*img_h+j)+2]=img_cv->at<Vec3b>(j,i)[c2]/255.0l;
            img_scr[(i*img_h+j)]=img_scr_cv->at<Vec3b>(j,i)[0]/255.0l;
        }
    }

    int size_vec_img=img_w*img_h*sizeof(double);
    hipMemcpy(dev_img, img, 3*size_vec_img, hipMemcpyHostToDevice);

    int n_warps=(img_w*img_h+31)/32;
    int n_blocos=n_mp_dev[0];
    int warps_pb=(n_warps+n_blocos-1)/n_blocos;

    while(warps_pb>MAX_WARPS){
        n_blocos+=n_mp_dev[0];
        warps_pb=(n_warps+n_blocos-1)/n_blocos;;
    }
        double epsilon=0.000001;
    loop<<<n_blocos,32*warps_pb>>>(dev_img, dev_row_inds, dev_col_inds, dev_vals, img_w, img_h,epsilon);

    hipMemcpy(h_row_inds, dev_row_inds, tlen*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_col_inds, dev_col_inds, tlen*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_vals, dev_vals, tlen*sizeof(double), hipMemcpyDeviceToHost);

    lambda=0.0001;
    VectorXd denom(img_w*img_h);
    #pragma omp parallel for
    for(int i=0; i<img_w*img_h; i++){
        denom(i)=img_scr[i]*lambda;
    }

    SparseMatrix<double> mat(img_w*img_h,img_w*img_h);

    std::vector<Triplet<double> > tripletList;
    tripletList.resize(tlen+img_w*img_h);
    #pragma omp parallel for
    for(int i=0; i<tlen; i++){
        tripletList[i]=Triplet<double>(h_row_inds[i],h_col_inds[i],h_vals[i]);
    }
    #pragma omp parallel for
    for(int i=0; i<img_w*img_h; i++){
        tripletList[tlen+i]=Triplet<double>(i,i,lambda);
    }

    printf("threads eigen %d\n", nbThreads());

    mat.setFromTriplets(tripletList.begin(), tripletList.end());
    printf("acabou\n");
    SimplicialCholesky<SparseMatrix<double> > chol(mat);
    printf("acabou2\n");
    VectorXd alpha = chol.solve(denom);
    h_alpha_mtrx=alpha.data();

    double alpha_min, alpha_max, img_min, img_max;
    alpha_min=h_alpha_mtrx[0];
    alpha_max=h_alpha_mtrx[0];
    img_min=img_scr[0];
    img_max=img_scr[0];
    #pragma omp parallel for
    for(int i=1; i<img_w*img_h; i++){
        if(h_alpha_mtrx[i]>alpha_max){
            alpha_max=h_alpha_mtrx[i];
        } else if(h_alpha_mtrx[i]<alpha_min){
            alpha_min=h_alpha_mtrx[i];
        }
        if(img_scr[i]>img_max){
            img_max=img_scr[i];
        } else if(img_scr[i]<img_min){
            img_min=img_scr[i];
        }
    }

    alpha_max-=alpha_min;
    #pragma omp parallel for
    for(int i=0; i<img_w*img_h; i++){
        h_alpha_mtrx[i]-=alpha_min;
        h_alpha_mtrx[i]=alpha[i]/alpha_max;
        h_alpha_mtrx[i]=alpha[i]*(img_max-img_min)+img_min;
    }

    Mat alpha_img(img_h, img_w, CV_8UC1);
    #pragma omp parallel for
    for(int i=0; i<img_w; i++){
        for(int j=0;j<img_h; j++){
            alpha_img.at<uchar>(j,i)=h_alpha_mtrx[i*img_h+j]*255;
        }
    }
    return alpha_img;
}

