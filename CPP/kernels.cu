
#include <hip/hip_runtime.h>
#define WINSIZE 1
const int nebsize=(WINSIZE*2+1)*(WINSIZE*2+1);

__global__ void loop(double *img, int *row_inds, int *col_inds, double *vals, int w, int h, double epsilon){
    int ind=blockIdx.x*blockDim.x+threadIdx.x;
    if(ind<w*h && ind%h>=WINSIZE && ind%h<h-WINSIZE && ind/h>=WINSIZE && ind/h<w-WINSIZE){
        int win_inds[nebsize];
        double tvals[nebsize*nebsize];
        int k;
        k=0;
        for(int i=-WINSIZE; i<=WINSIZE; i++){
            for(int j=-WINSIZE; j<=WINSIZE; j++){
                win_inds[k]=ind+i*h+j;
                k++;
            }
        }
        double winI[3*nebsize];
        double winI_aux[3*nebsize];
        double pre_win_var[9];
        double win_var[9];
        double win_mu[3];
        double detwin;
        k=0;
        for(int i=-WINSIZE; i<=WINSIZE; i++){
            memcpy(&winI[3*k*(WINSIZE*2+1)],&img[3*(ind+i*h-WINSIZE)], 3*(WINSIZE*2+1)*sizeof(double));
            k++;
        }

        win_mu[0]=0;
        win_mu[1]=0;
        win_mu[2]=0;
        for(int i=0; i<nebsize; i++){
            win_mu[0]+=winI[3*i];
            win_mu[1]+=winI[3*i+1];
            win_mu[2]+=winI[3*i+2];
        }
        win_mu[0]=win_mu[0]/nebsize;
        win_mu[1]=win_mu[1]/nebsize;
        win_mu[2]=win_mu[2]/nebsize;
        for(int i=0; i<3; i++){
            for(int j=0; j<3; j++){
                pre_win_var[3*i+j]=0;
                for(int n=0; n<nebsize; n++){
                    pre_win_var[3*i+j]+=winI[3*n+i]*winI[3*n+j];
                }
                pre_win_var[3*i+j]=pre_win_var[3*i+j]/nebsize;
                pre_win_var[3*i+j]+=(i==j)*epsilon/nebsize-win_mu[j]*win_mu[i];
            }
        }

        //inversa
        detwin=pre_win_var[0]*pre_win_var[4]*pre_win_var[8]+pre_win_var[2]*pre_win_var[3]*pre_win_var[7]+pre_win_var[1]*pre_win_var[5]*pre_win_var[6];
        detwin-=pre_win_var[6]*pre_win_var[4]*pre_win_var[2]+pre_win_var[3]*pre_win_var[1]*pre_win_var[8]+pre_win_var[7]*pre_win_var[5]*pre_win_var[0];

        win_var[0]=(pre_win_var[4]*pre_win_var[8]-pre_win_var[5]*pre_win_var[7])/detwin;
        win_var[3]=-(pre_win_var[3]*pre_win_var[8]-pre_win_var[5]*pre_win_var[6])/detwin;
        win_var[6]=(pre_win_var[3]*pre_win_var[7]-pre_win_var[4]*pre_win_var[6])/detwin;
        win_var[1]=-(pre_win_var[1]*pre_win_var[8]-pre_win_var[2]*pre_win_var[7])/detwin;
        win_var[4]=(pre_win_var[0]*pre_win_var[8]-pre_win_var[2]*pre_win_var[6])/detwin;
        win_var[7]=-(pre_win_var[0]*pre_win_var[7]-pre_win_var[1]*pre_win_var[6])/detwin;
        win_var[2]=(pre_win_var[1]*pre_win_var[5]-pre_win_var[2]*pre_win_var[4])/detwin;
        win_var[5]=-(pre_win_var[0]*pre_win_var[5]-pre_win_var[2]*pre_win_var[3])/detwin;
        win_var[8]=(pre_win_var[0]*pre_win_var[4]-pre_win_var[1]*pre_win_var[3])/detwin;
        //fim da inversa


        for(int i=0; i<nebsize; i++){
                winI[3*i+0]-=win_mu[0];
                winI[3*i+1]-=win_mu[1];
                winI[3*i+2]-=win_mu[2];
        }

        for(int i=0; i<nebsize; i++){
                for(int j=0; j<3; j++){
                    winI_aux[3*i+j]=0;
                    for(int n=0; n<3; n++){
                        winI_aux[3*i+j]+=winI[3*i+n]*win_var[3*n+j];
                    }
                }
        }

        for(int i=0; i<nebsize; i++){
            for(int j=0; j<nebsize; j++){
                tvals[i*nebsize+j]=0;
                for(int n=0; n<3; n++){
                    tvals[i*nebsize+j]+=winI_aux[3*i+n]*winI[3*j+n];
                }
                tvals[i*nebsize+j]++;
                tvals[i*nebsize+j]=(i==j)-tvals[i*nebsize+j]/nebsize;
            }
        }

        k=0;
        int base=((ind/h-WINSIZE)*(h-2*WINSIZE)+(ind%h-WINSIZE))*nebsize*nebsize;
        for(int i=0; i<nebsize; i++){
            for(int j=0; j<nebsize; j++){
                row_inds[base+k]=win_inds[j];
                col_inds[base+k]=win_inds[i];
                k++;
            }
        }

        memcpy(&vals[base],tvals, nebsize*nebsize*sizeof(double));
    }
}
